#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global void radix(int *a, int *b, int n, int count){
	int id = threadIdx.x;
	int i=0, data=0, j=0, pos=0;
	int temp = a[id];

	while(j<=count){
		data=temp%10;
		temp/=10;
		j++;
	}

	for(i=0; i<n; i++){
		j=0;
		int data2, temp=a[i];
		while(j<=count){
			data2=temp%10;
			temp/=10;
			j++;
		}
		if(data2<data || (data2==data && i<id))
			pos++;
	}
	b[pos]=a[id];
}

int main(){
	//Standard input
	int A[]={170,450,750,900,802,240,200,660};
	int n=8;
	int i, k=3; //3 digit

	int size = sizeof(int)*n;

	int *d_a, *d_b;

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);

	for(i=0; i<k; i++){
		hipMemcpy(d_a,&A,size,cudaMempyHostToDevice);
		radix<<<1,n>>>(d_a,d_b,n,i);
		hipMemcpy(&A,d_b,size, hipMemcpyDeviceToHost);
	}

	for(i=0; i<8; i++)
		printf("\n%d",A[i]);

	hipFree(d_a);
	hipFree(d_b);

	getchar()

	return 0;

}